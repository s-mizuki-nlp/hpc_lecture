
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void thread(float *a) {
  a[threadIdx.x] = threadIdx.x;
}

int main(void) {
  const int N = 4;
  // 共有メモリモデル
  // float *a;
  // cudaMallocManaged(&a, N*sizeof(float));

  // 分散メモリモデル
  float *a;
  float *b=(float*)malloc(N*sizeof(float));
  hipMalloc(&a,N*sizeof(float));
  // cudaMemcpy(b,a,N*sizeof(float),cudaMemcpyHostToDevice);

  thread<<<1,N>>>(a);
  // cudaDeviceSynchronize();
  hipMemcpy(a,b,N*sizeof(float),hipMemcpyDeviceToHost);
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,b[i]);
  hipFree(a);
}